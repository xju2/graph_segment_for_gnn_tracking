#include "hip/hip_runtime.h"
#include <cugraph/graph.hpp>
#include <cugraph/graph_functions.hpp>               // for weakly_connected_components, etc.
#include <cugraph/algorithms.hpp>                    // for BFS
#include <rmm/device_uvector.hpp>
#include <raft/handle.hpp>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <unordered_map>
#include <algorithm>
#include <chrono>

// Boost for reading DOT
#include <boost/graph/graphviz.hpp>
#include <boost/property_map/property_map.hpp>
#include <boost/graph/adjacency_list.hpp>

// -----------------------------------------------------------------------------
// Original "Boost Graph" definitions for CPU side only (parsing DOT):
// -----------------------------------------------------------------------------
typedef boost::property<boost::vertex_name_t, int64_t> vertex_p;
typedef boost::property<boost::edge_weight_t, double>  edge_p;

typedef boost::adjacency_list<
    boost::vecS, boost::vecS,
    boost::bidirectionalS,
    vertex_p,
    edge_p
> CPU_Graph;

// -----------------------------------------------------------------------------
// We'll define a struct for edges we keep after filtering (like "cleanup_graph")
// -----------------------------------------------------------------------------
struct EdgeData {
    int32_t src;
    int32_t dst;
    float   w;
};

// -----------------------------------------------------------------------------
// Example function to replicate logic in "get_tracks", but via cuGraph
// -----------------------------------------------------------------------------
std::vector<std::vector<int>> run_cugraph_pipeline(const CPU_Graph& G,
                                                   double cc_cut,
                                                   double th_min,
                                                   double th_add)
{
    // ---------------------------
    // 1) Build host-side adjacency from G, skipping edges <= cc_cut
    // ---------------------------
    int32_t nV = static_cast<int32_t>(boost::num_vertices(G));
    if (nV == 0) {
        return {};
    }
    int32_t nE_original = static_cast<int32_t>(boost::num_edges(G));

    // We'll store "hit ID" in a host array
    std::vector<int64_t> host_hit_id(nV);
    for (auto v_it = boost::vertices(G).first; v_it != boost::vertices(G).second; ++v_it) {
        int32_t v = static_cast<int32_t>(*v_it);
        int64_t name = boost::get(boost::vertex_name, G, *v_it);
        host_hit_id[v] = name;
    }

    // Collect edges, skipping low-weight
    std::vector<EdgeData> edges;
    edges.reserve(nE_original);

    for (auto e_it = boost::edges(G).first; e_it != boost::edges(G).second; ++e_it) {
        int32_t s = static_cast<int32_t>(boost::source(*e_it, G));
        int32_t t = static_cast<int32_t>(boost::target(*e_it, G));
        double w  = boost::get(boost::edge_weight, G, *e_it);
        if (w > cc_cut) {    // keep edge
            edges.push_back({ s, t, static_cast<float>(w) });
        }
    }
    // Edges are now a "cleaned" version.

    // ---------------------------
    // 2) Create cuGraph device structures
    //    We'll build an SG (single-GPU) graph for simplicity
    // ---------------------------
    raft::handle_t handle; // The RAFT handle for all rapids libs
    // We store source/dest in device vectors
    rmm::device_uvector<int32_t> d_src(edges.size(), handle.get_stream());
    rmm::device_uvector<int32_t> d_dst(edges.size(), handle.get_stream());
    rmm::device_uvector<float>   d_weights(edges.size(), handle.get_stream());

    // Copy edges to device
    for (size_t i = 0; i < edges.size(); ++i) {
        d_src.element(i) = edges[i].src;
        d_dst.element(i) = edges[i].dst;
        d_weights.element(i) = edges[i].w;
    }

    // We assume no multi-GPU, no renumbering needed if vertex IDs are 0..nV-1
    bool has_data = (edges.size() > 0);
    bool sorted_by_source = false;   // We didn't necessarily sort edges by source
    bool sorted_by_dest   = false;
    bool do_renumber      = false;   // If G was not guaranteed to have contiguous IDs, set true

    // Create a GraphCSR or GraphCOO if you prefer. 
    // We'll build an "edgelist_t" then call cugraph::create_graph.
    cugraph::edgelist_t<int32_t, int32_t, float> edgelist{
        d_src.data(),
        d_dst.data(),
        d_weights.data(),
        static_cast<int32_t>(edges.size())
    };

    // Graph properties
    cugraph::graph_properties_t graph_props{ /* is_symmetric = ???, etc.*/ };
    // For a bidirectionalS (like original) we do a directed = false approach if you want
    // to treat it as undirected. But let's say we want it directed:
    bool store_transposed = false; // Typically BFS on a forward graph

    auto graph_tuple = cugraph::create_graph<int32_t, int32_t, float, false, false>(
        handle,
        edgelist,
        cugraph::graph_meta_t<int32_t, int32_t>{ do_renumber, sorted_by_source, sorted_by_dest },
        nV,
        graph_props,
        store_transposed
    );

    // The returned tuple is: (unique_ptr<graph_t>, unique_ptr<resource_handle>, optional renumber_map)
    auto& graph_ptr = std::get<0>(graph_tuple); // graph_t
    auto& edge_weights_ptr = std::get<1>(graph_tuple); // optional edge_weights_t 
    // no renumber_map in get<2> if do_renumber=false

    auto graph_view = graph_ptr->view();
    auto weight_view = edge_weights_ptr ? edge_weights_ptr->view() : std::optional<rmm::device_uvector<float>>{};

    // ---------------------------
    // 3) Weakly Connected Components
    //    If you want an "undirected" sense of components
    // ---------------------------
    // For a directed graph, you might do strongly_connected_components. 
    // Let's do a "weakly" approach:
    rmm::device_uvector<int32_t> d_components(nV, handle.get_stream());

    bool connect_undirected = true; // treat edges as undirected
    cugraph::weakly_connected_components<int32_t, int32_t>(
        handle,
        graph_view,
        d_components.data(),
        connect_undirected
    );

    // bring them back to CPU
    std::vector<int32_t> h_components(nV);
    raft::update_host(h_components.data(), d_components.data(), nV, handle.get_stream());
    handle.sync_stream();

    // Group vertices by component
    std::unordered_map<int32_t, std::vector<int32_t>> comp_map;
    for (int32_t v = 0; v < nV; ++v) {
        comp_map[h_components[v]].push_back(v);
    }

    // We'll create a "sub_graphs" for the "simple paths" if deg <= 2, size >=3
    // We'll do degree check by calling cugraph::degree
    auto out_degs = cugraph::degree<int32_t>(handle, graph_view, cugraph::degree_direction::OUT); 
    // If you want in_degs + out_degs, you'd compute them or do an undirected approach:
    // out_degs has length nV. Let's also do an in_degs if needed:
    auto in_degs  = cugraph::degree<int32_t>(handle, graph_view, cugraph::degree_direction::IN);
    // We'll combine them for total deg in CPU
    std::vector<int32_t> h_out_degs(nV), h_in_degs(nV);
    raft::update_host(h_out_degs.data(), out_degs.data(), nV, handle.get_stream());
    raft::update_host(h_in_degs.data(), in_degs.data(), nV, handle.get_stream());
    handle.sync_stream();

    // Build "simple path" sub-graphs
    std::vector<std::vector<int>> sub_graphs;
    for (auto &kv : comp_map) {
        auto &compVec = kv.second;
        if ((int)compVec.size() < 3) continue;
        bool is_simple = true;
        for (auto vv : compVec) {
            // total deg = in + out
            int total_deg = h_out_degs[vv] + h_in_degs[vv];
            if (total_deg > 2) {
                is_simple = false;
                break;
            }
        }
        if (!is_simple) continue;
        // build final "track" of hit IDs
        std::vector<int> track;
        track.reserve(compVec.size());
        for (auto vv : compVec) {
            track.push_back(static_cast<int>(host_hit_id[vv]));
        }
        sub_graphs.push_back(std::move(track));
    }
    int num_simple_paths = (int) sub_graphs.size();

    // ---------------------------
    // 4) BFS example: a stand-in for "walkthrough"
    //    cugraph has BFS from a single source. For multiple, you'd loop or do multi-source BFS
    // ---------------------------
    // We'll do BFS from an arbitrary starting vertex. 
    // In your code, you might do multiple starts in reverse topological order, 
    // but cugraph doesn't have a built-in topological sort. 
    // We'll do a single BFS from vertex 0 for demonstration.
    rmm::device_uvector<int32_t> d_distances(nV, handle.get_stream());
    rmm::device_uvector<int32_t> d_predecessors(nV, handle.get_stream());

    // init BFS
    cugraph::bfs<int32_t, int32_t, float>(
        handle,
        graph_view,
        weight_view,        // optional edge weights
        d_distances.data(),
        d_predecessors.data(),
        0,                  // start from vertex 0
        false,              // direction_optimizing
        std::numeric_limits<int32_t>::max()
    );
    // BFS results are in d_distances, d_predecessors. We can build a path from 0 to each reachable vertex
    // We'll do a trivial example: any path of length >= 3 => add it to sub_graphs
    std::vector<int32_t> h_pred(nV);
    raft::update_host(h_pred.data(), d_predecessors.data(), nV, handle.get_stream());
    handle.sync_stream();

    // For demonstration: pick a vertex X, rebuild path back to 0
    int X = nV - 1; 
    std::vector<int> path;
    while (X != -1 && X < nV) {
        path.push_back((int)host_hit_id[X]);
        X = h_pred[X];
        if (X < 0) break;
    }
    if ((int)path.size() >= 3) {
        std::reverse(path.begin(), path.end()); 
        sub_graphs.push_back(std::move(path));
    }

    std::cout << "From CC&&Walk: BFS tracks found: " 
              << (sub_graphs.size() - num_simple_paths) << std::endl;

    return sub_graphs;
}

// -----------------------------------------------------------------------------
// Simple I/O for final track writing
// -----------------------------------------------------------------------------
void write_tracks(const std::vector<std::vector<int>>& tracks, const std::string& filename)
{
    std::ofstream file(filename);
    if (!file) {
        std::cerr << "Error: Unable to open " << filename << std::endl;
        return;
    }
    for (auto &t : tracks) {
        for (auto hid : t) {
            file << hid << " ";
        }
        file << "-1 ";
    }
    file << "\n";
}

// -----------------------------------------------------------------------------
// main
// -----------------------------------------------------------------------------
int main(int argc, char** argv)
{
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <dot_file>\n";
        return 1;
    }
    std::string dot_file_name(argv[1]);

    // -- CPU side: read the DOT file using Boost
    std::ifstream dot_file(dot_file_name);
    if (!dot_file) {
        std::cerr << "Error: Unable to open " << dot_file_name << std::endl;
        return 1;
    }

    CPU_Graph G;
    boost::dynamic_properties dp;
    dp.property("hit_id", boost::get(boost::vertex_name, G));
    dp.property("edge_scores", boost::get(boost::edge_weight, G));

    if (!boost::read_graphviz(dot_file, G, dp, "hit_id")) {
        std::cerr << "Error: Unable to parse graph from DOT file.\n";
        return 1;
    }
    double cc_cut = 0.01, th_min = 0.1, th_add = 0.6;

    // Time the pipeline
    auto start_t = std::chrono::high_resolution_clock::now();
    auto final_tracks = run_cugraph_pipeline(G, cc_cut, th_min, th_add);
    auto end_t = std::chrono::high_resolution_clock::now();
    double elapsed_ms = std::chrono::duration<double,std::milli>(end_t - start_t).count();

    std::cout << "Found " << final_tracks.size() << " total tracks.\n";
    std::cout << "Time (cuGraph) = " << elapsed_ms << " ms\n";

    // Write out results
    write_tracks(final_tracks, "tracks.txt");
    return 0;
}
